#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <vector>

#include <ros/ros.h>

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#include "ransac_cuda.hpp"

/**
 * @brief cudaにて直線検知を行う
 * 
 * @param urg             urgから得た点群データ
 * @param robotPos        機体の自己位置
 * @param warp            ブロックごとのthread間隔
 * @param fieldLine       fieldの木枠の直線データ
 * @param fieldFrame      木枠の角データ
 * @param pushDataLine    検知した直線データを格納
 * @param pushDataLineId  検知した直線データの種類
 * @param pushUrgData     x-y座標に直したurg点群データ
 */
void excuse_ransac_cuda(std::vector<tUrg> &urg, tPos robotPos, double warp,
                        std::vector<tLine> &fieldLine,
                        std::vector<tFrame> &fieldFrame,
                        std::vector<tLine> &pushDataLine,
                        std::vector<int> &pushDataLineId,
                        std::vector<tFrame> &pushUrgData)
{
  // if there in not enough points call return.
  if (urg.size() < 300)
  {
    std::cerr << "Error : urgFrame.size" << std::endl;
    return;
  }

  const int diffThresh = 60; //(int)((double)urg.size() / 5);   //鬪値.
  const int ransacMax = 720; //試行回数.

  // device上のメモリ定義.
  thrust::device_vector<tUrg> device_urg_data(urg.size());

  thrust::host_vector<tFrame> host_Frame_data(urg.size());
  thrust::device_vector<tFrame> device_Frame_data(urg.size());
  thrust::device_vector<int> device_Frame_id(urg.size());

  thrust::device_vector<tLine> device_field_data(fieldLine.size());
  thrust::device_vector<tFrame> device_field_frame(fieldFrame.size());

  // copy data host to  device.
  hipMemcpy(thrust::raw_pointer_cast(device_urg_data.data()), urg.data(),
             urg.size() * sizeof(tFrame), hipMemcpyHostToDevice);

  // copy data host to  device.
  hipMemcpy(thrust::raw_pointer_cast(device_field_data.data()),
             fieldLine.data(), fieldLine.size() * sizeof(tLine),
             hipMemcpyHostToDevice);

  // copy data host to  device.
  hipMemcpy(thrust::raw_pointer_cast(device_field_frame.data()),
             fieldFrame.data(), fieldFrame.size() * sizeof(tFrame),
             hipMemcpyHostToDevice);

  // detect area for program.
  dim3 blocks_0((int)urg.size() / warp + 1, 1, 1);
  dim3 threads_0((int)warp, 1, 1);

  // changr urg data
  urg_kernel<<<blocks_0, threads_0>>>(
      thrust::raw_pointer_cast(device_urg_data.data()), urg.size(),
      thrust::raw_pointer_cast(device_field_data.data()),
      device_field_data.size(),
      thrust::raw_pointer_cast(device_field_frame.data()), robotPos,
      thrust::raw_pointer_cast(device_Frame_data.data()),
      thrust::raw_pointer_cast(device_Frame_id.data()));
  　
　 //コピー
      hipMemcpy(thrust::raw_pointer_cast(host_Frame_data.data()),
                 thrust::raw_pointer_cast(device_Frame_data.data()),
                 device_Frame_data.size() * sizeof(tFrame), hipMemcpyDeviceToHost);

  for (int i = 0; i < urg.size(); i++)
    pushUrgData.push_back(host_Frame_data[i]);

  // gpuの領域設定.
  dim3 blocks_1((int)ransacMax / warp + 1, 1, 1);
  dim3 threads_1((int)warp, 1, 1);

  // ransacのデータのメモリ定義.
  thrust::host_vector<tLine> host_line_data(ransacMax);
  thrust::device_vector<tLine> device_line_data(ransacMax);
  thrust::host_vector<int> host_nearPoints_data(ransacMax);
  thrust::device_vector<int> device_nearPoints_data(ransacMax);
  thrust::host_vector<int> host_line_id_data(ransacMax);
  thrust::device_vector<int> device_line_id_data(ransacMax);

  // ransac.
  ransac_kernel<<<blocks_1, threads_1>>>(
      thrust::raw_pointer_cast(device_Frame_data.data()),
      thrust::raw_pointer_cast(device_Frame_id.data()),
      device_Frame_data.size(),
      thrust::raw_pointer_cast(device_line_data.data()),
      thrust::raw_pointer_cast(device_line_id_data.data()),
      thrust::raw_pointer_cast(device_nearPoints_data.data()), ransacMax);

  //コピー
  hipMemcpy(thrust::raw_pointer_cast(host_line_data.data()),
             thrust::raw_pointer_cast(device_line_data.data()),
             device_line_data.size() * sizeof(tLine), hipMemcpyDeviceToHost);

  hipMemcpy(thrust::raw_pointer_cast(host_nearPoints_data.data()),
             thrust::raw_pointer_cast(device_nearPoints_data.data()),
             device_nearPoints_data.size() * sizeof(int),
             hipMemcpyDeviceToHost);

  hipMemcpy(thrust::raw_pointer_cast(host_line_id_data.data()),
             thrust::raw_pointer_cast(device_line_id_data.data()),
             device_line_id_data.size() * sizeof(int), hipMemcpyDeviceToHost);

  //一番精度のいい直線を検出
  int max_diff[fieldLine.size()];
  int max_id[fieldLine.size()];
  for (int i = 0; i < fieldLine.size(); i++)
  {
    max_diff[i] = 0;
    max_id[i] = -1;
  }

  for (int i = 0; i < ransacMax; i++)
  {
    int tgtLineId = host_line_id_data[i];
    if (tgtLineId < 0)
      continue;
    if (host_nearPoints_data[i] > max_diff[tgtLineId])
    {
      max_diff[tgtLineId] = host_nearPoints_data[i];
      max_id[tgtLineId] = i;
    }
  }

  for (int i = 0; i < fieldLine.size(); i++)
  {
    if (max_diff[i] < diffThresh)
      continue;

    pushDataLine.push_back(host_line_data[max_id[i]]);
    pushDataLineId.push_back(i);
  }
}

/**
 * @brief 点群(length,angke)をfieldのx-y座標に変換
 * 
 * @param urg           urgから得た点群データ
 * @param urgNum 　　　　点群の数
 * @param fieldLine     fieldの直線
 * @param fieldLineNum  fieldの直線の数
 * @param fieldFrame    fieldの角のデータ
 * @param robotPos      機体の自己位置
 * @param out           x-y平面に直したもの
 * @param frame_id      点群に一番近い木枠のid
 */
__global__ void urg_kernel(tUrg *urg, int urgNum, tLine *fieldLine,
                           int fieldLineNum, tFrame *fieldFrame, tPos robotPos,
                           tFrame *out, int *frame_id)
{
  //threadのid
  int id = blockIdx.x * blockDim.x + threadIdx.x;

  //idは32の倍数にするのであまりが出る
  if (id > urgNum)
    return;

  //urgデータを機体中心座標に変換
  double x, y;
  x = -urg[id].length * sin(urg[id].angle);
  y = -urg[id].length * cos(urg[id].angle) - 370;

  //x = urg[id].length * cos(-urg[id].angle) + 370;
  //y = urg[id].length * sin(-urg[id].angle);

  //printf("%d, %lf, %lf\n", id, y, robotPos.angle);

  //機体中心座標からfield座標に変換
  out[id].x = x * cos(robotPos.angle) - y * sin(robotPos.angle) + robotPos.x;
  out[id].y = x * sin(robotPos.angle) + y * cos(robotPos.angle) + robotPos.y;

  //点群に一番近い木枠を決める
  double minLengthThresh = 400; //鬪値.
  int out_id = -1;
  for (int i = 0; i < fieldLineNum; i++)
  {
    double length =
        fabs(fieldLine[i].a * out[id].x + fieldLine[i].b * out[id].y +
             fieldLine[i].c) /
        sqrt(fieldLine[i].a * fieldLine[i].a + fieldLine[i].b * fieldLine[i].b);

    if (out[id].x < fieldFrame[i * 2].x - minLengthThresh ||
        out[id].x > fieldFrame[i * 2 + 1].x + minLengthThresh)
      continue;
    if (out[id].y < fieldFrame[i * 2].y - minLengthThresh ||
        out[id].y > fieldFrame[i * 2 + 1].y + minLengthThresh)
      continue;

    if (length < minLengthThresh)
    {
      minLengthThresh = length;
      out_id = i;
    }
  }
  frame_id[id] = out_id;
}

/**
 * @brief ransac(直線検知を実行)
 * 
 * @param urg             urgの点群データ
 * @param urg_id          点群がどこの木枠に所属してるか計算
 * @param urgNum          点群の数
 * @param line            直線のパラメータ
 * @param nearPointsId    直線がどの木枠に所属してるか
 * @param nearPointsNum 　直線に近い点群の数
 * @param ransacMax       ransacの施行回数
 */
__global__ void ransac_kernel(tFrame *urg, int *urg_id, int urgNum, tLine *line,
                              int *nearPointsId, int *nearPointsNum,
                              int ransacMax)
{
  // idを計算.
  int id = blockIdx.x * blockDim.x + threadIdx.x;
  if (id > ransacMax)
    return;

  // cuda版rand関数
  hiprandState_t state;
  hiprand_init(clock(), id, 0, &state);

  // ransacに必要な変数.
  const double lengthThresh = 20; //鬪値.
  const int sampleNum = 2;        // sample数.

  // randamな位置を取得.
  unsigned int randam[sampleNum]; // randamな値.
  tFrame randamPoints[sampleNum]; // ramdamな位置.
  int tgtLineId = -1;

  for (int i = 0; i < sampleNum; i++)
  {
    // randamな値を入手.
    randam[i] = hiprand(&state) % urgNum;

    randamPoints[i].x = urg[randam[i]].x;
    randamPoints[i].y = urg[randam[i]].y;
  }

  if (urg_id[randam[0]] == -1 || urg_id[randam[1]] == -1)
  {
    nearPointsId[id] = -1;
    nearPointsNum[id] = 0;
    return;
  }

  if (urg_id[randam[0]] != urg_id[randam[1]])
  {
    nearPointsId[id] = -1;
    nearPointsNum[id] = 0;
    return;
  }

  tgtLineId = urg_id[randam[0]];

  // ax + by + c = 0;(2点の場合).
  //※３点の場合最小２乗法
  tLine randamLine;
  randamLine.a = randamPoints[1].y - randamPoints[0].y;
  randamLine.b = -(randamPoints[1].x - randamPoints[0].x);
  randamLine.c = -randamPoints[1].y * randamPoints[0].x +
                 randamPoints[1].x * randamPoints[0].y;

  //最小２乗法で直線計算(ax + by + c = 0).
  //double xySum = 0, xSum = 0, ySum = 0, xPowSum = 0;
  int near = 0;
  for (int i = 0; i < urgNum; i++)
  {
    //直線と点の距離を計算.
    double length =
        fabs(randamLine.a * urg[i].x + randamLine.b * urg[i].y + randamLine.c) /
        sqrt(randamLine.a * randamLine.a + randamLine.b * randamLine.b);
    if (length < lengthThresh)
    {
      //直線と点の距離を計算し鬪値以内ならpointを足す.
      near++;

      //最小２乗法.
      /*
      xySum += urg[i].x * urg[i].y / 1000000;
      xSum += urg[i].x / 1000;
      ySum += urg[i].y / 1000;
      xPowSum += urg[i].x * urg[i].x / 1000000;
      */
    }
  }

  //最小２乗法.
  /*
  tLine leastLine;
  leastLine.a = near * xySum - xSum * ySum;
  leastLine.b = xSum * xSum - near * xPowSum;
  leastLine.c = xPowSum * ySum - xySum * xSum;
  if (leastLine.a > leastLine.b) {
    leastLine.b /= leastLine.a;
    leastLine.c /= leastLine.a / 1000;
    leastLine.a = 1;
  }
  else {
    leastLine.a /= leastLine.b;
    leastLine.c /= leastLine.b / 1000;
    leastLine.b = 1;
  }*/

  //line[id] = leastLine;
  line[id] = randamLine;
  nearPointsNum[id] = near;
  nearPointsId[id] = tgtLineId;
}
